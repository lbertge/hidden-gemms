#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <chrono>
#include <assert.h>
#include <random>

// Tile size for shared memory
#define TILE_WIDTH 32

// Naive kernel implementation
__global__ void naiveMatrixMulKernel(float* A, float* B, float* C, 
                                    int M, int N, int K,
                                    float alpha, float beta) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    // const int row = blockIdx.x * TILE_WIDTH + (threadIdx.x / TILE_WIDTH);
    // const int col = blockIdx.y * TILE_WIDTH + (threadIdx.x % TILE_WIDTH);

    
    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = alpha * sum + beta * C[row * N + col];
    }
}

// Naive wrapper function
void naiveMatrixMul(float* A, float* B, float* C, 
                    int M, int N, int K,
                    float alpha, float beta) {
    dim3 blockDim(32, 32);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, 
                 (M + blockDim.y - 1) / blockDim.y);
    
    naiveMatrixMulKernel<<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
}

// Benchmark function
void benchmark(int M, int N, int K, int num_iterations = 10) {
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);
    
    // Allocate host memory
    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];
    float *h_C_cublas = new float[M * N];
    
    // Initialize matrices
    int some_seed = 759;
    std::mt19937 generator(some_seed);
    std::uniform_real_distribution<float> distribution(0.0f, 1.0f);
    for (int i = 0; i < M * K; i++) h_A[i] = distribution(generator);
    for (int i = 0; i < K * N; i++) h_B[i] = distribution(generator);
    for (int i = 0; i < M * N; i++) h_C[i] = distribution(generator);
    
    // Allocate device memory
    float *d_A, *d_B, *d_C, *d_C_cublas;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);
    hipMalloc(&d_C_cublas, size_C);
    
    // Copy data to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice);
    hipMemcpy(d_C_cublas, h_C, size_C, hipMemcpyHostToDevice);
    
    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    // Constants for SGEMM
    float alpha = 0.5f;
    float beta = 0.5f;
    
    // Warmup
    naiveMatrixMul(d_A, d_B, d_C, M, N, K, alpha, beta);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                N, M, K, &alpha,
                d_B, N, d_A, K, &beta,
                d_C_cublas, N);
    
    // Benchmark naive implementation
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    float naive_time;
    hipEventRecord(start);
    for (int i = 0; i < num_iterations; i++) {
        naiveMatrixMul(d_A, d_B, d_C, M, N, K, alpha, beta);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&naive_time, start, stop);
    naive_time /= num_iterations;
    
    // Benchmark cuBLAS
    hipEventRecord(start);
    for (int i = 0; i < num_iterations; i++) {
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    N, M, K, &alpha,
                    d_B, N, d_A, K, &beta,
                    d_C_cublas, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float cublas_time;
    hipEventElapsedTime(&cublas_time, start, stop);
    cublas_time /= num_iterations;
    
    // Calculate GFLOPS
    double operations = 2.0 * M * N * K;  // multiply-adds
    double naive_gflops = (operations * 1e-9) / (naive_time * 1e-3);
    double cublas_gflops = (operations * 1e-9) / (cublas_time * 1e-3);
    
    // Print results
    printf("Matrix dimensions: M=%d, N=%d, K=%d\n", M, N, K);
    printf("Naive implementation: %.3f ms (%.2f GFLOP/s)\n", naive_time, naive_gflops);
    printf("cuBLAS implementation: %.3f ms (%.2f GFLOP/s)\n", cublas_time, cublas_gflops);
    printf("Performance ratio:\n");
    printf("  cuBLAS/naive: %.2fx\n", naive_time/cublas_time);
    
    // Verify results
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);
    hipMemcpy(h_C_cublas, d_C_cublas, size_C, hipMemcpyDeviceToHost);
    
    // Check results
    float epsilon = 1e-3;
    for (int i = 0; i < M * N; i++) {
        assert(fabs(h_C[i] - h_C_cublas[i]) < epsilon);
    }
    
    // Cleanup
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_cublas;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_C_cublas);
    hipblasDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    // Test with different matrix sizes
    int sizes[] = {512, 1024, 2048, 4096};
    
    for (int size : sizes) {
        benchmark(size, size, size);
        printf("\n");
    }
    
    return 0;
} 