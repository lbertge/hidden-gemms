#include "hip/hip_runtime.h"
#include "../src/host.cuh"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <chrono>
#include <assert.h>
#include <random>

// Updated benchmark function
void benchmark(int M, int N, int K, int num_iterations = 10) {
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);
    
    // Allocate host memory
    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];
    float *h_C_cublas = new float[M * N];
    
    // Initialize matrices
    int some_seed = 759;
    std::mt19937 generator(some_seed);
    std::uniform_real_distribution<float> distribution(-10.0f, 10.0f);
    for (int i = 0; i < M * K; i++) h_A[i] = distribution(generator);
    for (int i = 0; i < K * N; i++) h_B[i] = distribution(generator);
    for (int i = 0; i < M * N; i++) h_C[i] = distribution(generator);  // Initialize C as well
    
    // Allocate device memory
    float *d_A, *d_B, *d_C, *d_C_cublas;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);
    hipMalloc(&d_C_cublas, size_C);
    
    // Copy data to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice);
    hipMemcpy(d_C_cublas, h_C, size_C, hipMemcpyHostToDevice);  // Copy same C data
    
    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    // Constants for SGEMM
    float alpha = 0.5f;
    float beta = 0.5f;
    
    // Warmup
    shared_memory_host(d_A, d_B, d_C, M, N, K, alpha, beta);
    cublas_host(d_A, d_B, d_C_cublas, M, N, K, alpha, beta, handle);
    
    // Benchmark shared memory implementation
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    for (int i = 0; i < num_iterations; i++) {
        shared_memory_host(d_A, d_B, d_C, M, N, K, alpha, beta);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float custom_time;
    hipEventElapsedTime(&custom_time, start, stop);
    custom_time /= num_iterations;
    
    // Benchmark cuBLAS
    hipEventRecord(start);
    for (int i = 0; i < num_iterations; i++) {
        cublas_host(d_A, d_B, d_C_cublas, M, N, K, alpha, beta, handle);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float cublas_time;
    hipEventElapsedTime(&cublas_time, start, stop);
    cublas_time /= num_iterations;
    
    // Calculate GFLOPS
    // For matrix multiplication: 2*M*N*K operations (M*N*K multiplications and M*N*K additions)
    double operations = 2.0 * M * N * K;
    double gflops = (operations * 1e-9) / (custom_time * 1e-3); // Convert ms to s
    double cublas_gflops = (operations * 1e-9) / (cublas_time * 1e-3);
    
    // Print results
    printf("Matrix dimensions: M=%d, N=%d, K=%d\n", M, N, K);
    printf("Custom implementation: %.3f ms (%.2f GFLOP/s)\n", custom_time, gflops);
    printf("cuBLAS implementation: %.3f ms (%.2f GFLOP/s)\n", cublas_time, cublas_gflops);
    printf("Performance ratio (cuBLAS/custom): %.2fx\n", custom_time/cublas_time);
    
    // Verify results
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);
    hipMemcpy(h_C_cublas, d_C_cublas, size_C, hipMemcpyDeviceToHost);
    
    // assert that the results are within some epsilon of each other
    float epsilon = 1e-3;
    for (int i = 0; i < M * N; i++) {
        assert(fabs(h_C[i] - h_C_cublas[i]) < epsilon);
    }

    // Cleanup
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_cublas;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_C_cublas);
    hipblasDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    // Test with different matrix sizes
    int sizes[] = {512, 1024, 2048, 4096};
    
    for (int size : sizes) {
        benchmark(size, size, size);
        printf("\n");
    }
    
    return 0;
}