#include "hip/hip_runtime.h"
#include "../src/host.cuh"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <chrono>
#include <assert.h>
#include <random>

// Benchmark function
void benchmark(int M, int N, int K, int num_iterations = 10) {
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);
    
    // Allocate host memory
    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];
    float *h_C_cublas = new float[M * N];
    
    // Initialize matrices
    int some_seed = 759;
    std::mt19937 generator(some_seed);
    std::uniform_real_distribution<float> distribution(0.0f, 1.0f);
    for (int i = 0; i < M * K; i++) h_A[i] = distribution(generator);
    for (int i = 0; i < K * N; i++) h_B[i] = distribution(generator);
    for (int i = 0; i < M * N; i++) h_C[i] = distribution(generator);
    
    // Allocate device memory
    float *d_A, *d_B, *d_C, *d_C_cublas;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);
    hipMalloc(&d_C_cublas, size_C);
    
    // Copy data to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice);
    hipMemcpy(d_C_cublas, h_C, size_C, hipMemcpyHostToDevice);
    
    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    // Constants for SGEMM
    float alpha = 0.5f;
    float beta = 0.5f;
    
    // Warmup
    coal_host(d_A, d_B, d_C, M, N, K, alpha, beta);
    cublas_host(d_A, d_B, d_C_cublas, M, N, K, alpha, beta, handle);
    
    // Benchmark naive implementation
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    float coal_ram_time;
    hipEventRecord(start);
    for (int i = 0; i < num_iterations; i++) {
        coal_host(d_A, d_B, d_C, M, N, K, alpha, beta);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&coal_ram_time, start, stop);
    coal_ram_time /= num_iterations;
    
    // Benchmark cuBLAS
    hipEventRecord(start);
    for (int i = 0; i < num_iterations; i++) {
        cublas_host(d_A, d_B, d_C_cublas, M, N, K, alpha, beta, handle);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float cublas_time;
    hipEventElapsedTime(&cublas_time, start, stop);
    cublas_time /= num_iterations;
    
    // Calculate GFLOPS
    double operations = 2.0 * M * N * K;  // multiply-adds
    double naive_gflops = (operations * 1e-9) / (coal_ram_time * 1e-3);
    double cublas_gflops = (operations * 1e-9) / (cublas_time * 1e-3);
    
    // Print results
    printf("Matrix dimensions: M=%d, N=%d, K=%d\n", M, N, K);
    printf("Coal Ram implementation: %.3f ms (%.2f GFLOP/s)\n", coal_ram_time, naive_gflops);
    printf("cuBLAS implementation: %.3f ms (%.2f GFLOP/s)\n", cublas_time, cublas_gflops);
    printf("Performance ratio:\n");
    printf("  cuBLAS/coal_ram: %.2fx\n", coal_ram_time/cublas_time);
    
    // Verify results
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);
    hipMemcpy(h_C_cublas, d_C_cublas, size_C, hipMemcpyDeviceToHost);
    
    // Check results
    float epsilon = 1e-2;
    for (int i = 0; i < M * N; i++) {
        assert(fabs(h_C[i] - h_C_cublas[i]) < epsilon);
    }
    
    // Cleanup
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_cublas;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_C_cublas);
    hipblasDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    // Test with different matrix sizes
    int sizes[] = {512, 1024, 2048, 4096};
    
    for (int size : sizes) {
        benchmark(size, size, size);
        printf("\n");
    }
    
    return 0;
} 