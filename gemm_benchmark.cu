#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <chrono>

// Tile size for shared memory
#define TILE_WIDTH 32

// CUDA kernel for tiled matrix multiplication
__global__ void tiledMatrixMulKernel(float* A, float* B, float* C, 
                                    int M, int N, int K) {
    __shared__ float As[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // Calculate row and column for this thread
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;
    
    float sum = 0.0f;
    
    // Loop over tiles
    for (int t = 0; t < (K + TILE_WIDTH - 1) / TILE_WIDTH; t++) {
        // Load tiles into shared memory
        if (row < M && t * TILE_WIDTH + tx < K)
            As[ty][tx] = A[row * K + t * TILE_WIDTH + tx];
        else
            As[ty][tx] = 0.0f;
            
        if (t * TILE_WIDTH + ty < K && col < N)
            Bs[ty][tx] = B[(t * TILE_WIDTH + ty) * N + col];
        else
            Bs[ty][tx] = 0.0f;
            
        __syncthreads();
        
        // Compute partial dot product
        for (int k = 0; k < TILE_WIDTH; k++) {
            sum += As[ty][k] * Bs[k][tx];
        }
        
        __syncthreads();
    }
    
    // Write result
    if (row < M && col < N)
        C[row * N + col] = sum;
}

// Wrapper function for kernel launch
void tiledMatrixMul(float* A, float* B, float* C, int M, int N, int K) {
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid((N + TILE_WIDTH - 1) / TILE_WIDTH, 
                 (M + TILE_WIDTH - 1) / TILE_WIDTH);
    
    tiledMatrixMulKernel<<<dimGrid, dimBlock>>>(A, B, C, M, N, K);
}

// Benchmark function
void benchmark(int M, int N, int K, int num_iterations = 10) {
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);
    
    // Allocate host memory
    float *h_A = (float*)malloc(size_A);
    float *h_B = (float*)malloc(size_B);
    float *h_C = (float*)malloc(size_C);
    float *h_C_cublas = (float*)malloc(size_C);
    
    // Initialize matrices
    for (int i = 0; i < M * K; i++) h_A[i] = rand() / (float)RAND_MAX;
    for (int i = 0; i < K * N; i++) h_B[i] = rand() / (float)RAND_MAX;
    
    // Allocate device memory
    float *d_A, *d_B, *d_C, *d_C_cublas;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);
    hipMalloc(&d_C_cublas, size_C);
    
    // Copy data to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    
    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    // Constants for hipblasSgemm
    float alpha = 1.0f;
    float beta = 0.0f;
    
    // Warmup
    tiledMatrixMul(d_A, d_B, d_C, M, N, K);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                N, M, K, &alpha,
                d_B, N, d_A, K, &beta,
                d_C_cublas, N);
    
    // Benchmark custom implementation
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    for (int i = 0; i < num_iterations; i++) {
        tiledMatrixMul(d_A, d_B, d_C, M, N, K);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float custom_time;
    hipEventElapsedTime(&custom_time, start, stop);
    custom_time /= num_iterations;
    
    // Benchmark cuBLAS
    hipEventRecord(start);
    for (int i = 0; i < num_iterations; i++) {
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    N, M, K, &alpha,
                    d_B, N, d_A, K, &beta,
                    d_C_cublas, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float cublas_time;
    hipEventElapsedTime(&cublas_time, start, stop);
    cublas_time /= num_iterations;
    
    // Print results
    printf("Matrix dimensions: M=%d, N=%d, K=%d\n", M, N, K);
    printf("Tiling implementation: %.3f ms\n", custom_time);
    printf("cuBLAS implementation: %.3f ms\n", cublas_time);
    printf("Performance ratio (cuBLAS/tiling): %.2fx\n", custom_time/cublas_time);
    
    // Verify results
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);
    hipMemcpy(h_C_cublas, d_C_cublas, size_C, hipMemcpyDeviceToHost);
    
    float max_diff = 0.0f;
    for (int i = 0; i < M * N; i++) {
        float diff = fabs(h_C[i] - h_C_cublas[i]);
        max_diff = max(max_diff, diff);
    }
    printf("Maximum difference from cuBLAS: %e\n", max_diff);
    
    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_cublas);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_C_cublas);
    hipblasDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    // Test with different matrix sizes
    int sizes[] = {512, 1024, 2048, 4096};
    
    for (int size : sizes) {
        benchmark(size, size, size);
        printf("\n");
    }
    
    return 0;
}