#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <hipblas.h>
#include <cmath> // For fabs
#include <iostream>

#define TILE_WIDTH 32
#define CEIL(M, N) (((M) + (N) - 1) / (N))

#define EPSILON 1e-1

bool compare_results(const float* kernel, const float* cublas, int M, int N) {
    bool match = true;
    for (int i = 0; i < M * N; ++i) {
        if (std::fabs(kernel[i] - cublas[i]) > EPSILON) {
            match = false;
            std::cout << "Mismatch at index " << i 
                      << " (row " << i / N << ", col " << i % N << "): "
                      << "Kernel result = " << kernel[i] 
                      << ", Cublas result = " << cublas[i] 
                      << ", diff = " << std::fabs(kernel[i] - cublas[i]) 
                      << '\n';
        }
    }
    return match;
}

// Naive host
void naive_host(float* A, float* B, float* C, int M, int N, int K, float alpha, float beta) {
    dim3 gridDim(CEIL(M, TILE_WIDTH), CEIL(N, TILE_WIDTH));
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
    
    naive_kernel<<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
}

// CoalRam host
void coal_host(float* A, float* B, float* C, int M, int N, int K, float alpha, float beta) {
    dim3 gridDim(CEIL(M, TILE_WIDTH), CEIL(N, TILE_WIDTH));
    dim3 blockDim(TILE_WIDTH * TILE_WIDTH);
    
    coal_kernel<<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
}

// Shared Memory host
void shared_memory_host(float* A, float* B, float* C, int M, int N, int K, float alpha, float beta) {
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(CEIL(M, TILE_WIDTH), CEIL(N, TILE_WIDTH));
    
    shared_memory_kernel<<<dimGrid, dimBlock>>>(A, B, C, M, N, K, alpha, beta);
}

// Tiled Matrix Multiplication 1D host
void block_tiling_1d_host(float* A, float* B, float* C, int M, int N, int K, float alpha, float beta) {
    // Block 1d tiling parameters
    const int BM = 64;
    const int BN = 64;
    const int BK = 8;
    const int TM = 8;
    dim3 gridDim(CEIL(M, BM), CEIL(N, BN));
    dim3 blockDim(BM / TM * BN);
    block_tiling_1d_kernel<BM, BN, BK, TM><<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
}


// Tiled Matrix Multiplication 2D host
void block_tiling_2d_host(float* A, float* B, float* C, int M, int N, int K, float alpha, float beta) {
    // Block 2d tiling parameters
    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;
    dim3 gridDim(CEIL(M, BM), CEIL(N, BN));
    dim3 blockDim(BM / TM * BN / TN);
    block_tiling_2d_kernel<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
}


// Vectorized host
void vectorized_host(float* A, float* B, float* C, int M, int N, int K, float alpha, float beta) {
    // Block 2d tiling parameters
    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;
    dim3 gridDim(CEIL(M, BM), CEIL(N, BN));
    dim3 blockDim(BM / TM * BN / TN);
    vectorized_kernel<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
}


// Double Buffered host
void double_buffered_host(float* A, float* B, float* C, int M, int N, int K, float alpha, float beta) {
    // Block 2d tiling parameters
    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;
    dim3 gridDim(CEIL(M, BM), CEIL(N, BN));
    dim3 blockDim(BM / TM * BN / TN);
    double_buffered_kernel<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
}


// cuBLAS host
void cublas_host(float* A, float* B, float* C, int M, int N, int K, float alpha, float beta, hipblasHandle_t handle) {
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, N, A, K, &beta, C, N);
}

void cpu_gemm(float *A, float *B, float *C, int M, int N, int K, float alpha, float beta) {
    // Scale matrix C by beta
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            C[i * N + j] *= beta;
        }
    }

    // Perform matrix multiplication
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < K; ++k) {
                sum += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] += alpha * sum;
        }
    }
}